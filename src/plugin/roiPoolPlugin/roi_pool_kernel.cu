#include "hip/hip_runtime.h"
#include <stdio.h>

#include <algorithm>
#include <cmath>

#include "amir_cuda_util/cuda_util.h"
#include "roi_pool.h"

namespace amirstan {
namespace plugin {
using namespace amirstan::cuda;
const int kMAX_FEATMAP_SIZE = 10;
struct FeatData {
    const void* data[kMAX_FEATMAP_SIZE];
    int         batch_size;
    int         channels;
    int         h[kMAX_FEATMAP_SIZE];
    int         w[kMAX_FEATMAP_SIZE];
    float       spatial_scale[kMAX_FEATMAP_SIZE];
    int         num_featmap;
};

template<typename scalar_t>
__device__ scalar_t
bilinear_interpolate(const scalar_t* bottom_data, const int height, const int width, scalar_t y, scalar_t x)
{
    // deal with cases that inverse elements are out of feature map boundary
    if (y < -1.0 || y > height || x < -1.0 || x > width) {
        return 0;
    }

    if (y <= 0)
        y = 0;
    if (x <= 0)
        x = 0;

    int y_low = (int)y;
    int x_low = (int)x;
    int y_high;
    int x_high;

    if (y_low >= height - 1) {
        y_high = y_low = height - 1;
        y              = (scalar_t)y_low;
    }
    else {
        y_high = y_low + 1;
    }

    if (x_low >= width - 1) {
        x_high = x_low = width - 1;
        x              = (scalar_t)x_low;
    }
    else {
        x_high = x_low + 1;
    }

    scalar_t ly = y - y_low;
    scalar_t lx = x - x_low;
    scalar_t hy = 1. - ly;
    scalar_t hx = 1. - lx;
    // do bilinear interpolation
    scalar_t lt = bottom_data[y_low * width + x_low];
    scalar_t rt = bottom_data[y_low * width + x_high];
    scalar_t lb = bottom_data[y_high * width + x_low];
    scalar_t rb = bottom_data[y_high * width + x_high];
    scalar_t w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

    scalar_t val = (w1 * lt + w2 * rt + w3 * lb + w4 * rb);

    return val;
}

template<typename scalar_t>
__device__ scalar_t roi_pool_single(const scalar_t* bottom_data,
                                    const int       roi_batch_ind,
                                    const scalar_t  roi_start_w,
                                    const scalar_t  roi_start_h,
                                    const scalar_t  roi_end_w,
                                    const scalar_t  roi_end_h,
                                    const scalar_t  spatial_scale,
                                    const int       pw,
                                    const int       ph,
                                    const int       c,
                                    const int       channels,
                                    const int       height,
                                    const int       width,
                                    const int       pooled_height,
                                    const int       pooled_width)
{
    // Force malformed ROIs to be 1x1
    scalar_t roi_width  = fmaxf((scalar_t)roi_end_w - (scalar_t)roi_start_w, 0.);
    scalar_t roi_height = fmaxf((scalar_t)roi_end_h - (scalar_t)roi_start_h, 0.);
    if (roi_width < 1e-5 && roi_height < 1e-5) {
        return 0.;
    }

    const scalar_t bin_size_h = roi_height / (scalar_t)pooled_height;
    const scalar_t bin_size_w = roi_width / (scalar_t)pooled_width;

    // the corresponding bin region
    int bin_x1 = floor(static_cast<scalar_t>(pw) * bin_size_w + roi_start_w);
    int bin_y1 = floor(static_cast<scalar_t>(ph) * bin_size_h + roi_start_h);
    int bin_x2 = ceil(static_cast<scalar_t>(pw + 1) * bin_size_w + roi_start_w);
    int bin_y2 = ceil(static_cast<scalar_t>(ph + 1) * bin_size_h + roi_start_h);

    // add roi offsets and clip to input boundaries
    bin_x1        = min(max(bin_x1, 0), width);
    bin_y1        = min(max(bin_y1, 0), height);
    bin_x2        = min(max(bin_x2, 0), width);
    bin_y2        = min(max(bin_y2, 0), height);
    bool is_empty = (bin_y2 <= bin_y1) || (bin_x2 <= bin_x1);

    const scalar_t* offset_bottom_data = bottom_data + (roi_batch_ind * channels + c) * height * width;

    scalar_t max_val = is_empty ? 0 : -1e10;
    for (int h = bin_y1; h < bin_y2; ++h) {
        for (int w = bin_x1; w < bin_x2; ++w) {
            int offset = h * width + w;
            if (offset_bottom_data[offset] > max_val) {
                max_val = offset_bottom_data[offset];
            }
        }
    }

    return max_val;
}

template<typename scalar_t>
__global__ void roi_pool_kernel(scalar_t*       output,
                                const scalar_t* bottom_rois,
                                FeatData        feat_data,
                                const float     roi_scale_factor,
                                const int       finest_scale,
                                const int       pooled_height,
                                const int       pooled_width,
                                int             nThreads)
{
    CUDA_KERNEL_LOOP(index, nThreads)
    {
        const int channels = feat_data.channels;
        const int pw       = index % pooled_width;
        const int ph       = (index / pooled_width) % pooled_height;
        const int c        = (index / pooled_width / pooled_height) % channels;
        const int n        = index / pooled_width / pooled_height / channels;

        const scalar_t* offset_bottom_rois = bottom_rois + n * 5;

        scalar_t roi_offset_x0 = offset_bottom_rois[1];
        scalar_t roi_offset_y0 = offset_bottom_rois[2];
        scalar_t roi_offset_x1 = offset_bottom_rois[3];
        scalar_t roi_offset_y1 = offset_bottom_rois[4];

        const scalar_t scale = sqrtf((roi_offset_y1 - roi_offset_y0 + 1.) * (roi_offset_x1 - roi_offset_x0 + 1.));

        const int target_lvls =
            fminf(feat_data.num_featmap - 1, fmaxf(0, floorf(log2f(scale / (scalar_t)(finest_scale) + 1e-6))));

        if (roi_scale_factor > 0.) {
            const scalar_t roi_off_cx = (roi_offset_x0 + roi_offset_x1) * 0.5;
            const scalar_t roi_off_cy = (roi_offset_y0 + roi_offset_y1) * 0.5;
            const scalar_t roi_off_w  = (roi_offset_x1 - roi_offset_x0 + 1) * roi_scale_factor;
            const scalar_t roi_off_h  = (roi_offset_y1 - roi_offset_y0 + 1) * roi_scale_factor;

            roi_offset_x0 = roi_off_cx - roi_off_w * 0.5 + 0.5;
            roi_offset_x1 = roi_off_cx + roi_off_w * 0.5 - 0.5;
            roi_offset_y0 = roi_off_cy - roi_off_h * 0.5 + 0.5;
            roi_offset_y1 = roi_off_cy + roi_off_h * 0.5 - 0.5;
        }

        const scalar_t  spatial_scale = (scalar_t)feat_data.spatial_scale[target_lvls];
        const int       height        = feat_data.h[target_lvls];
        const int       width         = feat_data.w[target_lvls];
        const scalar_t* bottom_data   = (scalar_t*)feat_data.data[target_lvls];

        const int      roi_batch_ind = offset_bottom_rois[0];
        const scalar_t roi_start_w   = roi_offset_x0 * spatial_scale;
        const scalar_t roi_start_h   = roi_offset_y0 * spatial_scale;
        const scalar_t roi_end_w     = (roi_offset_x1 + 1) * spatial_scale;
        const scalar_t roi_end_h     = (roi_offset_y1 + 1) * spatial_scale;

        const scalar_t output_val = roi_pool_single<scalar_t>(bottom_data,
                                                              roi_batch_ind,
                                                              roi_start_w,
                                                              roi_start_h,
                                                              roi_end_w,
                                                              roi_end_h,
                                                              spatial_scale,
                                                              pw,
                                                              ph,
                                                              c,
                                                              channels,
                                                              height,
                                                              width,
                                                              pooled_height,
                                                              pooled_width);

        output[index] = output_val;
    }
}

template<typename T>
void roi_pool(T*                 output,
              const T*           rois,
              int                num_rois,
              const void* const* feats,
              int                num_feats,
              int                n,
              int                c,
              int*               h,
              int*               w,
              float*             strides,
              int                out_size,
              float              roi_scale_factor,
              int                finest_scale,
              hipStream_t       stream)
{
    FeatData feat_data;
    feat_data.batch_size  = n;
    feat_data.channels    = c;
    feat_data.num_featmap = num_feats;
    for (int i = 0; i < num_feats; ++i) {
        feat_data.data[i]          = feats[i];
        feat_data.h[i]             = h[i];
        feat_data.w[i]             = w[i];
        feat_data.spatial_scale[i] = 1. / float(strides[i]);
    }
    int pooled_height = out_size;
    int pooled_width  = out_size;
    int nThreads      = num_rois * c * pooled_height * pooled_width;

    // bool aligned = true;
    roi_pool_kernel<T><<<GET_BLOCKS(nThreads), CUDA_NUM_THREADS, 0, stream>>>(
        output, rois, feat_data, roi_scale_factor, finest_scale, pooled_height, pooled_width, nThreads);
}

template void roi_pool<float>(float*             output,
                              const float*       rois,
                              int                num_rois,
                              const void* const* feats,
                              int                num_feats,
                              int                n,
                              int                c,
                              int*               h,
                              int*               w,
                              float*             strides,
                              int                out_size,
                              float              roi_scale_factor,
                              int                finest_scale,
                              hipStream_t       stream);

}  // namespace plugin
}  // namespace amirstan
