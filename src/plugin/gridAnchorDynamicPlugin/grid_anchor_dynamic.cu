#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <stdio.h>

#include <algorithm>
#include <cmath>

#include "amir_cuda_util/cuda_util.h"
#include "grid_anchor_dynamic.h"

namespace amirstan {
namespace plugin {
using namespace amirstan::cuda;
template<typename T>
__global__ void grid_anchor_dynamic_kernel(
    T* __restrict__ output, const T* __restrict__ base_anchor, int width, int height, int stride, int num_base_anchor)
{
    CUDA_KERNEL_LOOP(i, width * height * num_base_anchor)
    {
        const int y       = i / (width * num_base_anchor);
        const int x       = (i % (width * num_base_anchor)) / num_base_anchor;
        const int base_id = i % num_base_anchor;

        output[i * 4 + 0] = base_anchor[base_id * 4 + 0] + x * stride;
        output[i * 4 + 1] = base_anchor[base_id * 4 + 1] + y * stride;
        output[i * 4 + 2] = base_anchor[base_id * 4 + 2] + x * stride;
        output[i * 4 + 3] = base_anchor[base_id * 4 + 3] + y * stride;
    }
}

template<>
__global__ void grid_anchor_dynamic_kernel<float>(float* __restrict__ output,
                                                  const float* __restrict__ base_anchor,
                                                  int width,
                                                  int height,
                                                  int stride,
                                                  int num_base_anchor)
{
    CUDA_KERNEL_LOOP(i, width * height * num_base_anchor)
    {
        const int y       = i / (width * num_base_anchor);
        const int x       = (i % (width * num_base_anchor)) / num_base_anchor;
        const int base_id = i % num_base_anchor;

        const float4 base_anchor_val = reinterpret_cast<const float4*>(base_anchor)[base_id];

        const float stride_f = float(stride);
        const float x_f      = float(x);
        const float y_f      = float(y);
        float4      out_val;
        out_val.x = fma(x_f, stride_f, base_anchor_val.x);
        out_val.y = fma(y_f, stride_f, base_anchor_val.y);
        out_val.z = fma(x_f, stride_f, base_anchor_val.z);
        out_val.w = fma(y_f, stride_f, base_anchor_val.w);

        reinterpret_cast<float4*>(output)[i] = out_val;
    }
}

template<typename T>
void grid_anchor_dynamic(
    T* output, const T* base_anchor, int width, int height, int stride, int num_base_anchor, hipStream_t stream)
{
    size_t input_size = num_base_anchor * height * width;
    grid_anchor_dynamic_kernel<T><<<GET_BLOCKS(input_size), CUDA_NUM_THREADS, 0, stream>>>(
        output, base_anchor, width, height, stride, num_base_anchor);
}

template void grid_anchor_dynamic<float>(float*       output,
                                         const float* base_anchor,
                                         int          width,
                                         int          height,
                                         int          stride,
                                         int          num_base_anchor,
                                         hipStream_t stream);

}  // namespace plugin
}  // namespace amirstan
