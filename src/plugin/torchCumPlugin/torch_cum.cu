#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <stdio.h>

#include <algorithm>
#include <cmath>
#include <hipcub/hipcub.hpp>

#include "amir_cuda_util/cuda_util.h"
#include "torch_cum.h"

namespace amirstan {
namespace plugin {
using namespace amirstan::cuda;

template<typename T>
struct CumScanProd {
    __host__ __device__ __forceinline__ T operator()(const T& a, const T& b)
    {
        return a * b;
    }
};

template<typename T>
__global__ void
torch_cum_warp_kernel(T* output, const T* input, size_t stride, int dim_size, size_t cum_size, const int cum_type)
{
    // create block scan
    typedef hipcub::WarpScan<T> warpScan;
    __shared__ union {
        typename warpScan::TempStorage scan[CUDA_NUM_WARP];
    } temp_storage;

    for (int index = (blockIdx.x * CUDA_NUM_WARP) + int(threadIdx.x / CUDA_WARP_SIZE); index < cum_size;
         index += gridDim.x * CUDA_NUM_WARP) {
        // compute cum start
        const size_t pre_index  = index / stride;
        const size_t post_index = index % stride;

        const size_t cum_start = pre_index * stride * dim_size + post_index;

        T aggregate_value = (T)0;
        if (cum_type == 1) {
            aggregate_value = (T)1;
        }

        for (int warp_offset = 0; warp_offset < dim_size; warp_offset += CUDA_WARP_SIZE) {
            const size_t cum_position = warp_offset + threadIdx.x % CUDA_WARP_SIZE;
            T            thread_data  = cum_position < dim_size ? input[cum_start + cum_position * stride] : 0;
            if (cum_type == 0) {
                if (threadIdx.x % CUDA_WARP_SIZE == 0) {
                    thread_data = thread_data + aggregate_value;
                }
                warpScan(temp_storage.scan[int(threadIdx.x / CUDA_WARP_SIZE)])
                    .InclusiveSum(thread_data, thread_data, aggregate_value);
            }
            else {
                if (threadIdx.x % CUDA_WARP_SIZE == 0) {
                    thread_data = thread_data * aggregate_value;
                }
                warpScan(temp_storage.scan[int(threadIdx.x / CUDA_WARP_SIZE)])
                    .InclusiveScan(thread_data, thread_data, CumScanProd<T>(), aggregate_value);
            }

            // Store scanned items to output segment
            if (cum_position < dim_size) {
                output[cum_start + cum_position * stride] = thread_data;
            }
        }
    }
}

static void create_size_stride(const int* dims, int nb_dims, TensorSize& size, TensorStride& stride)
{
    memcpy(&size.size[0], dims, sizeof(int) * nb_dims);
    stride.size[nb_dims - 1] = 1;
    for (int i = nb_dims - 2; i >= 0; --i) {
        stride.size[i] = stride.size[i + 1] * size.size[i + 1];
    }
}

template<typename T>
void torch_cum(T* output, const T* input, int* input_dims, int nb_dims, int cum_dim, int cum_type, hipStream_t stream)
{
    TensorSize   ts_input_size;
    TensorStride input_stride;
    create_size_stride(input_dims, nb_dims, ts_input_size, input_stride);

    size_t cum_size = 1;
    for (int i = 0; i < nb_dims; ++i) {
        if (i != cum_dim) {
            cum_size *= ts_input_size.size[i];
        }
    }

    size_t num_blocks = std::min<long>(kMaxGridNum, (cum_size + CUDA_NUM_WARP - 1) / CUDA_NUM_WARP);
    torch_cum_warp_kernel<T><<<num_blocks, CUDA_NUM_THREADS, 0, stream>>>(
        output, input, input_stride.size[cum_dim], ts_input_size.size[cum_dim], cum_size, cum_type);
}

template void torch_cum<float>(
    float* output, const float* input, int* input_dims, int nb_dims, int cum_dim, int cum_type, hipStream_t stream);

template void torch_cum<int>(
    int* output, const int* input, int* input_dims, int nb_dims, int cum_dim, int cum_type, hipStream_t stream);

}  // namespace plugin
}  // namespace amirstan
